#include "hip/hip_runtime.h"
#include "data_augmentation.cuh"

// Global random number generator and distributions for generating random numbers on the host. The random number generator used
// is the Mersenne Twister mt19937 from the BOOST library.
boost::random::mt19937 rng;
boost::random::normal_distribution<> snorm(0.0, 1.0); // Standard normal distribution
boost::random::uniform_real_distribution<> uniform(0.0, 1.0); // Uniform distribution from 0.0 to 1.0


// Initialize the parallel random number generator state on the device
__global__ void initialize_rng(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
     number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}
