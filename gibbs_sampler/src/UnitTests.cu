#include "hip/hip_runtime.h"
/*
 * UnitTests.cpp
 *
 *  Created on: Jul 17, 2013
 *      Author: brandonkelly
 */

// standard includes
#include <iostream>

// local includes
#include "UnitTests.cuh"
#include "data_augmentation.cuh"

// Global random number generator and distributions for generating random numbers on the host. The random number generator used
// is the Mersenne Twister mt19937 from the BOOST library. These are instantiated in data_augmentation.cu.
extern boost::random::mt19937 rng;
extern boost::random::normal_distribution<> snorm; // Standard normal distribution
extern boost::random::uniform_real_distribution<> uniform; // Uniform distribution from 0.0 to 1.0

// calculate transpose(x) * covar_inv * x for an nx-element vector x and an (nx,nx)-element matrix covar_inv
double mahalanobis_distance(double** covar_inv, double* x, int nx) {
	double distance = 0.0;
	double* x_temp;
	x_temp = new double [nx];
	for (int i = 0; i < nx; ++i) {
		x_temp[i] = 0.0;
		for (int j = 0; j < nx; ++j) {
			x_temp[i] += covar_inv[i][j] * x[j];
		}
	}
	for (int i = 0; i < nx; ++i) {
		distance += x_temp[i] * x[i];
	}
	delete x_temp;
	return distance;
}

double mean(double* x, int nx) {
	double mu = 0.0;
	for (int i = 0; i < nx; ++i) {
		mu += x[i] / nx;
	}
	return mu;
}

double variance(double* x, int nx) {
	double sigsqr = 0.0;
	double mu = mean(x, nx);
	for (int i = 0; i < nx; ++i) {
		sigsqr += (x[i] - mu) * (x[i] - mu);
	}
	sigsqr /= nx;
	return sigsqr;
}

// destructor
UnitTests::~UnitTests() {
	// free memory used by data arrays
	for (int i = 0; i < ndata; ++i) {
		delete [] meas[i];
		delete [] meas_unc[i];
	}
	delete meas;
	delete meas_unc;
}

// test rank-1 cholesky update
void UnitTests::R1CholUpdate() {
	std::cout << "Testing rank-1 Cholesky Update..." << std::endl;
	double covar[3][3] =
	{
			{5.29, 0.3105, -15.41},
			{0.3105, 0.2025, 3.2562},
			{-15.41, 3.2562, 179.56}
	};
	double cholfact0[6] = {2.3, 0.135, 0.42927264, -6.7, 9.6924416, 6.38173768};
	double v[3] = {-0.39108095, -0.0668706, -0.30427621};
	// cholesky factor for update (covar + v * transpose(v)), computed from python
	double Lup0[6] = {2.33301185, 0.14429923, 0.43145036, -6.55419017, 9.78632116,  6.39711602};
	// cholesky factor for downdate (covar - v * transpose(v)), computed from python
	double Ldown0[6] = {2.26650738, 0.12545654, 0.42695313, -6.85150942, 9.59219932,  6.36505672};

	// now get rank-1 updated and downdated factors using the fast method
	double Lup[6] = {2.3, 0.135, 0.42927264, -6.7, 9.6924416, 6.38173768};
	double vup[3] = {-0.39108095, -0.0668706, -0.30427621};
	bool downdate = false;
	chol_update_r1(Lup, vup, 3, downdate);

	double Ldown[6] = {2.3, 0.135, 0.42927264, -6.7, 9.6924416, 6.38173768};
	double vdown[3] = {-0.39108095, -0.0668706, -0.30427621};
	downdate = true;
	chol_update_r1(Ldown, vdown, 3, downdate);

	// test if the cholesky factors agree by finding the maximum fraction difference between the two
	double max_frac_diff_up = 0.0;
	double max_frac_diff_down = 0.0;
	for (int i = 0; i < 6; ++i) {
		double frac_diff = abs(Lup[i] - Lup0[i]) / abs(Lup0[i]);
		max_frac_diff_up = max(frac_diff, max_frac_diff_up);
		frac_diff = abs(Ldown[i] - Ldown0[i]) / abs(Ldown0[i]);
		max_frac_diff_down = max(frac_diff, max_frac_diff_down);
	}

	if ((max_frac_diff_down < epsilon) && (max_frac_diff_up < epsilon)) {
		// cholesky factors agree, so test passed
		npassed++;
	} else {
		// test failed
		std::cerr << "Rank-1 Cholesky update test failed." << std::endl;
	}

	nperformed++;
}

// check that Chi::Propose follows a multivariate normal distribution
void UnitTests::ChiPropose() {
	std::cout << "Testing Chi::Propose..." << std::endl;
	double covar[3][3] =
	{
			{5.29, 0.3105, -15.41},
			{0.3105, 0.2025, 3.2562},
			{-15.41, 3.2562, 179.56}
	};
	double covar_inv0[3][3] =
	{
			{0.64880351, -2.66823952, 0.10406763},
			{-2.66823952, 17.94430089, -0.55439855},
			{0.10406763, -0.55439855, 0.02455399}
	};
	double** covar_inv;
	covar_inv = new double* [3];
	for (int i = 0; i < 3; ++i) {
		covar_inv[i] = new double [3];
	}
	for (int i = 0; i < 3; ++i) {
		for (int j = 0; j < 3; ++j) {
			covar_inv[i][j] = covar_inv0[i][j];
		}
	}

	double cholfact[6] = {2.3, 0.135, 0.42927264, -6.7, 9.6924416, 6.38173768};

	int p = 3, ntrials = 100000;
	int m = 1, dt = 1, current_iter = 1;

	Characteristic Chi(p, m, dt, current_iter);
	Chi.SetRNG(&rng);

	double snorm_deviate[p];
	double scaled_proposal[p];
	double proposed_chi[p];
	double chi[3] = {1.2, 0.4, -0.7};
	double chisqr[ntrials];

	for (int i = 0; i < ntrials; ++i) {
		// get the ntrials proposals
		Chi.Propose(chi, cholfact, proposed_chi, snorm_deviate, scaled_proposal);
		for (int j = 0; j < p; ++j) {
			proposed_chi[j] -= chi[j]; // center the proposals
		}
		chisqr[i] = mahalanobis_distance(covar_inv, proposed_chi, p);
	}
	/*
	 * check that the values of chisqr are consistent with being drawn from a chi-square distribution
	 * with p = 3 degrees of freedom.
	*/

	// first compare the average with the known value
	double true_mean = p;
	double true_var = 2.0 * p;
	double mu_sigma = sqrt(true_var / ntrials); // standard deviation in the average
	double mean_chisqr = mean(chisqr, ntrials);

	double zdiff_mean = abs(mean_chisqr - true_mean) / mu_sigma;
	if (zdiff_mean < 3.0) {
		npassed++;
	} else {
		std::cerr << "Test for Chi::Propose failed: average chi-square value more than 3-sigma away from true value" << std::endl;
	}
	nperformed++;

	// compare empirical quantiles with known ones
	double chi2_low = 0.3, chi2_high = 8.0;
	int nlow_low = 3800, nlow_high = 4200; // # of chisqr < chi2_low should fall within this interval
	int nhigh_low = 95200, nhigh_high = 95600; // # of chisqr < chi2_high should fall within this interval
	int count_low = 0, count_high = 0;
	for (int i = 0; i < ntrials; ++i) {
		// count the number of elements of chisqr that are below the 4.0 and 95.4 percentiles
		if (chisqr[i] < chi2_low) {
			count_low++;
		}
		if (chisqr[i] < chi2_high) {
			count_high++;
		}
	}
	if ((count_low > nlow_low) && (count_low < nlow_high)) {
		npassed++;
	} else {
		std::cerr << "Test for Chi::Propose failed: empirical 4.0 percentile inconsistent with true value" << std::endl;
	}
	nperformed++;
	if ((count_high > nhigh_low) && (count_high < nhigh_high)) {
		npassed++;
	} else {
		std::cerr << "Test for Chi::Propose failed: empirical 95.4 percentile inconsistent with true value" << std::endl;
	}
	nperformed++;

	// free memory
	for (int i = 0; i < 3; ++i) {
		delete [] covar_inv[i];
	}
	delete covar_inv;
}

// check that Chi::Accept always accepts when the proposal and the current values are the same
void UnitTests::ChiAcceptSame() {
	double chi[3] = {1.2, 0.4, -0.7};
	int p = 3, ntrials = 100000;
	int m = 1, dt = 1, current_iter = 1;

	Characteristic Chi(p, m, dt, current_iter);
	Chi.SetRNG(&rng);

	bool accept;
	int naccept = 0;
	double logdens = -1.32456;
	double ratio = 0.0;
	for (int i = 0; i < ntrials; ++i) {
		accept = Chi.AcceptProp(logdens, logdens, 0.0, 0.0, ratio);
		if (abs(ratio - 1.0) < epsilon) {
			naccept++;
		}
	}

	if (naccept == ntrials) {
		npassed++;
	} else {
		std::cerr << "Test for Chi::Accept failed: Failes to always accept when the log-posteriors are the same." << std::endl;
	}
	nperformed++;
}

// test Chi::Adapt acceptance rate and covariance by running a simple MCMC sampler
void UnitTests::ChiAdapt() {

}

// check that PopulationPar::Propose follow a multivariate normal distribution
void UnitTests::ThetaPropose() {

}

// check that PopulationPar::Accept always accepts when the logdensities are the same
void UnitTests::ThetaAcceptSame() {

}

// make sure we accept and save a population parameter value with a much higher posterior
void UnitTests::ThetaAcceptBetter() {

}

// test PopulationPar::Adapt acceptance rate and covariance by running a simple MCMC sampler
void UnitTests::ThetaAdapt() {

}

// check that constructor for population parameter correctly set the pointer data member of DataAugmentation
void UnitTests::DaugPopPtr() {

}

// test DataAugmentation::GetChi
void UnitTests::DaugGetChi() {

}

// check that DataAugmentation::Update always accepts when the proposed and current chi values are the same
void UnitTests::DaugAcceptSame() {

}

// make sure that DataAugmentation::Update() accepts and saves Chi values when the posterior is much higher
void UnitTests::DaugAcceptBetter() {

}

// print out summary of test results
void UnitTests::Finish() {
	std::cout << npassed << " tests passed out of " << nperformed << " tests performed." << std::endl;
	npassed = 0;
	nperformed = 0;
}
