#include "hip/hip_runtime.h"
/*
 * cudahm_blueprint.cu
 *
 *  Created on: Mar 13, 2014
 *      Author: Brandon C. Kelly
 *
 * This file provides a blueprint for using the CUDAHM API. In order to use CUDAHM the user must supply a function
 * to compute the logarithm of the conditional probability densities of the measurements (y) given the
 * characteristics (chi), and the characteristics given the population parameter theta. The user must also set
 * the pointers c_pLogDensMeas and c_pLogDensPop to these functions, in order to correctly place these functions
 * in GPU constant memory. The purpose of this file is to provide an easy way of setting the pointers and stream-line
 * the use of CUDAHM to build MCMC sampler. Using this blueprint, the user need only modify the LogDensityMeas and
 * LogDensityPop functions to ensure that the pointers are set correctly.
 *
 * The MCMC sampler should be constructed within the main function using the GibbSampler class. Further details
 * are provided below. A complete working example is provided in normnorm.cu.
 *
 */

// standard library includes
#include <iostream>

// local CUDAHM includes
#include "GibbsSampler.hpp"
#include "input_output.hpp"
#include "ConstBetaTemp.cuh"
#include "DustPopPar.hpp"

/*
 * First you need to set the values of the parameter dimensions as const int types. These must be supplied
 * at compile-time in order to efficiently make use of GPU memory. These also need to be placed before the
 * functions LogDensityMeas and LogDensityPop if they need to know the dimensions of the features and parameters.
 *
 */

const int mfeat = 5;
const int pchi = 3;  // chi = {log C, beta, log T}, where C \propto N_H
const int dtheta = 9;
__constant__ const double c_nu[mfeat] = {6.0e11, 8.571e11, 1.2e11, 1.765e12, 4.286e12};  // {500, 350, 250, 170, 70} microns, Herschel bands
const double nu_ref = 2.3e11;  // 230 GHz
__constant__ double c_nu_ref = nu_ref;

const int dof = 8;  // population-level model is a multivariate student's t-distribution with dof degrees of freedom
__constant__ int c_dof = dof;

// physical constants, cgs
const double clight = 2.99792458e10;
__constant__ double c_clight = clight;
const double hplanck = 6.6260755e-27;
__constant__ double c_hplanck = hplanck;
const double kboltz = 1.380658e-16;
__constant__ double c_kboltz = kboltz;

// Compute the model dust SED, a modified blackbody
__device__
double modified_blackbody(double nu, double C, double beta, double T) {
	double sed = 2.0 * c_hplanck * nu * nu * nu / (c_clight * c_clight) / (exp(c_hplanck * nu / (c_kboltz * T)) - 1.0);
	sed *= C * pow(nu / c_nu_ref, beta);
	return sed;
}

/*
 * This function returns the logarithm of the conditional density of the measurements given the
 * characteristics for a single data point, log p(y_i | chi_i). This function must be supplied by the user
 * and written in CUDA. The input parameters are:
 *
 * 	chi      - Pointer to the values of the characteristics for the i^th data point.
 * 	meas     - Pointer to the measurements for the i^th data point.
 * 	meas_unc - Pointer to the standard deviations in the measurement errors for y_ij.
 * 	mfeat    - The number of features measured for each data point, i.e., the length of the meas array.
 *  pchi     - The number of characteristics for each data point, i.e., the length of the chi array.
 *
 */

__device__
double LogDensityMeas(double* chi, double* meas, double* meas_unc)
{
	double C = exp(chi[0]);
	double T = exp(chi[2]);
	double logdens_meas = 0.0;
	for (int j = 0; j < mfeat; ++j) {
		// p(y_ij | chi_ij) is a normal density centered at the model SED
		double model_sed = modified_blackbody(c_nu[j], C, chi[1], T);
		logdens_meas += -0.5 * (meas[j] - model_sed) * (meas[j] - model_sed) / (meas_unc[j] * meas_unc[j]);
	}
	return logdens_meas;
}

/*
 * Helper functions used by the function that computes the log-density of log C, beta, log T | theta
 */

// calculate the inverse of a 3 x 3 matrix
__device__ __host__
double matrix_invert3d(double* A, double* A_inv) {
	double determ_inv = 0.0;
	determ_inv = 1.0 / (A[0] * (A[4] * A[8] - A[5] * A[7]) - A[1] * (A[8] * A[3] - A[5] * A[6]) +
			A[2] * (A[3] * A[7] - A[4] * A[6]));

	A_inv[0] = determ_inv * (A[4] * A[8] - A[5] * A[7]);
	A_inv[1] = -determ_inv * (A[1] * A[8] - A[2] * A[7]);
	A_inv[2] = determ_inv * (A[1] * A[5]- A[2] * A[4]);
	A_inv[3] = -determ_inv * (A[3] * A[8] - A[5] * A[6]);
	A_inv[4] = determ_inv * (A[0] * A[8] - A[2] * A[6]);
	A_inv[5] = -determ_inv * (A[0] * A[5] - A[2] * A[3]);
	A_inv[6] = determ_inv * (A[3] * A[7] - A[4] * A[6]);
	A_inv[7] = -determ_inv * (A[0] * A[7] - A[1] * A[6]);
	A_inv[8] = determ_inv * (A[0] * A[4] - A[1] * A[3]);

	return determ_inv;
}

// calculate transpose(x) * covar_inv * x
__device__ __host__
double chisqr(double* x, double* covar_inv, int nx)
{
	double chisqr = 0.0;
	for (int i = 0; i < nx; ++i) {
		for (int j = 0; j < nx; ++j) {
			chisqr += x[i] * covar_inv[i * nx + j] * x[j];
		}
	}
	return chisqr;
}

//__device__ __host__
//double tanh(double x) {
//	return (exp(2.0 * x) - 1.0) / (exp(2.0 * x) + 1.0);
//}

/*
 * This function returns the logarithm of the conditional density of the characteristics given the
 * population parameter theta for a single data point, log p(chi_i | theta). This function must be supplied by
 * the user and written in CUDA. The input parameters are:
 *
 * 	chi       - Pointer to the values of the characteristics for the i^th data point.
 * 	theta     - Pointer to the population parameter.
 *  pchi      - The number of characteristics for each data point, i.e., the length of the chi array.
 *  dim_theta - The dimension of the population parameter vector theta, i.e., the length of the theta array.
 *
 */
__device__
double LogDensityPop(double* chi, double* theta)
{
	double covar[pchi * pchi];
	double covar_inv[pchi * pchi];
	double cov_determ_inv;

	// transform theta values to covariance matrix of (log C, beta, log T)
	covar[0] = exp(2.0 * theta[pchi]);  // Covar[0,0], variance in log C
	covar[1] = tanh(theta[2 * pchi]) * exp(theta[pchi] + theta[pchi+1]);  // Covar[0,1] = cov(log C, beta)
	covar[2] = tanh(theta[2 * pchi + 1]) * exp(theta[pchi] + theta[pchi+2]);  // Covar[0,2] = cov(log C, log T)
	covar[3] = covar[1];  // Covar[1,0]
	covar[4] = exp(2.0 * theta[pchi + 1]);  // Covar[1,1], variance in beta
	covar[5] = tanh(theta[2 * pchi + 2]) * exp(theta[pchi+1] + theta[pchi+2]);  // Covar[1,2] = cov(beta, log T)
	covar[6] = covar[2];  // Covar[2,0]
	covar[7] = covar[5];  // Covar[2,1]
	covar[8] = exp(2.0 * theta[pchi + 2]);  // Covar[2,2], variance in log T

	cov_determ_inv = matrix_invert3d(covar, covar_inv);
	double chi_cent[pchi];
	for (int j = 0; j < pchi; ++j) {
		chi_cent[j] = chi[j] - theta[j];
	}
	double zsqr = chisqr(chi_cent, covar_inv, pchi);

	// multivariate student's t-distribution with DOF degrees of freedom
	double logdens_pop = 0.5 * log(cov_determ_inv) - (pchi + c_dof) / 2.0 * log(1.0 + zsqr / pchi);

	return logdens_pop;
}

/*
 * Pointers to the GPU functions used to compute the conditional log-densities for a single data point.
 * These functions live on the GPU in constant memory.
 *
 * IF YOU ARE USING LogDensityMeas and LogDensityPop TO COMPUTE YOUR CONDITIONAL DENSITIES, DO NOT MODIFY THESE.
 * Otherwise you will need to set these points to whichever functions you are using to compute these quantities.
 *
 */
__constant__ pLogDensMeas c_LogDensMeas = LogDensityMeas;  // log p(y_i|chi_i)
__constant__ pLogDensPop c_LogDensPop = LogDensityPop;  // log p(chi_i|theta)

/*
 * Pointer to the population parameter (theta), stored in constant memory on the GPU. Originally defined in
 * kernels.cu and kernels.cuh. Needed by LogDensityPop, which computes the conditional posterior of the
 * characteristics given the population parameters: log p(chi_i|theta). This assumes a maximum of 100 elements
 * in the theta parameter vector.
 *
 * If you need more than this, you will have to change this manually here and in
 * the kernels.cuh and kernels.cu files.
 *
 * YOU SHOULD NOT MODIFY THIS UNLESS YOU KNOW WHAT YOU ARE DOING.
 */
extern __constant__ double c_theta[100];


int main(int argc, char** argv)
{
	/*
	 * Read in the data for the measurements, meas, and their standard deviations, meas_unc.
	 */

	std::string datafile = "../data/cbt_sed_1000.dat";
	int ndata = get_file_lines(datafile);
	std::cout << "Loaded " << ndata << " data points." << std::endl;

	vecvec fnu(ndata);
	vecvec fnu_sig(ndata);
	read_data(datafile, fnu, fnu_sig, ndata, mfeat);

	/*
	 * Set the number of MCMC iterations and the amount of thinning for the chi and theta samples.
	 *
	 * NOTE THAT IF YOU HAVE A LARGE DATA SET, YOU WILL PROBABLY WANT TO THIN THE CHI VALUES SIGNIFICANTLY SO
	 * YOU DO NOT RUN OUR OF MEMORY.
	 */

	int nmcmc_iter = 10;
	// int nburnin = nmcmc_iter / 2;
	int nburnin = 1;
	int nchi_samples = 10;
	int nthin_chi = nmcmc_iter / nchi_samples;

	/*
	 * Instantiate the GibbsSampler<mfeat, pchi, dtheta> object here. Once you've instantiated it, use the
	 * GibbSampler::Run() method to run the MCMC sampler. Finally, use GibbSampler::GetCharSampler() to get
	 * the sampled characteristics as a std::vector<std::vector<std::vector<double> > > (three nested vectors,
	 * dimensions nchi_samples x ndata x pchi) object. Similarly, use the GibbsSampler::GetPopSamples() to
	 * retrieve the sampled theta values as a std::vector<std::vector<double> > (two nested vectors,
	 * dimensions nsamples x dtheta) object.
	 */

	// first instantiate the subclassed DataAugmentation and PopulationPar objects
	boost::shared_ptr<DataAugmentation<mfeat, pchi, dtheta> > CBT(new ConstBetaTemp<mfeat, pchi, dtheta>(fnu, fnu_sig));
	boost::shared_ptr<PopulationPar<mfeat, pchi, dtheta> > Theta(new DustPopPar<mfeat, pchi, dtheta>);

	// instantiate the GibbsSampler object and run the sampler
	GibbsSampler<mfeat, pchi, dtheta> Sampler(CBT, Theta, nmcmc_iter, nburnin, nthin_chi);
	Sampler.Run();

   // grab the samples
	vecvec theta_samples = Sampler.GetPopSamples();
	std::vector<vecvec> chi_samples = Sampler.GetCharSamples();

	std::cout << "Writing results to text files..." << std::endl;

	// write the sampled theta values to a file. Output will have nsamples rows and dtheta columns.
	std::string thetafile("dusthm_thetas.dat");
	write_thetas(thetafile, theta_samples);

	// write the posterior means and standard deviations of the characteristics to a file. output will have ndata rows and
	// 2 * pchi columns, where the column format is posterior mean 1, posterior sigma 1, posterior mean 2, posterior sigma 2, etc.
	std::string chifile("dusthm_chi_summary.dat");
	write_chis(chifile, chi_samples);

}

