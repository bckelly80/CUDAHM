#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>

#include <stdio.h>
#include <fstream>
#include <stdlib.h>

//#include <thrust/sort.h>
//#include <thrust/copy.h>

//#include <thrust/transform_reduce.h>
//#include <thrust/functional.h>

//#include <algorithm>
//#include <cstdlib>

#include <vector>
/*
 typedef thrust::device_vector<double> dvec;
 typedef std::vector<dvec> vdvec;
 
 struct wrapvec
 {
 vdvec v;
 
 wrapvec(int m, int n) : v(m)
 {
 for (int i=0; i<m; i++)	{
 v[i].reserve(n);
 }
 }
 
 double** ptrs()
 {
 thrust::host_vector<double*> h_ptr(v.size());
 for (unsigned int i=0; i<v.size(); i++)
 h_ptr[i] = (double*) thrust::raw_pointer_cast(&(v[i][0]));
 
 thrust::device_vector<double*> d_ptr = h_ptr;
 return (double**) thrust::raw_pointer_cast(&d_ptr[0]);
 }
 };
 */
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)

// Target acceptance rate for Robust Adaptive Metropolis (RAM).
static __constant__ double c_target_rate = 0.4;

// Decay rate for proposal scaling factor updates. The proposal scaling factors decay as 1 / niter^decay_rate.
// This is gamma in the notation of Vihola (2012)
static __constant__ double c_decay_rate = 0.66667;

// Pointer to the current value of theta. This is stored in constant memory so that all the threads on the GPU
// can access the same theta quickly.
static __constant__ double c_theta[2];

// Initialize the random number generator state
__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
     number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

// Perform the MHA update on the n parameters, done in parallel on the GPU. This is what the kernel does.
__global__
void update_chi(double* chi, double* meas, double* meas_unc, int n, double* logdens, double* jump_sigma,
                hiprandState* state, int current_iter)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<n)
	{
        double mu = c_theta[0];  // Get population parameters
        double var = c_theta[1];
        
        double meas_i = meas[i];  // Get measurements
        double meas_unc_i = meas_unc[i];
        
        /* Copy state to local memory for efficiency */
        hiprandState localState = state[i];
        
        // Propose a new value of the characteristics for this data point
        double new_chi = chi[i] + jump_sigma[i] * hiprand_normal_double(&localState);

        // Compute the conditional log-posterior of the proposed parameter values for this data point
        double logdens_prop = -0.5 * (meas_i - new_chi) * (meas_i - new_chi) / (meas_unc_i * meas_unc_i) +
            0.5 * log(var) - 0.5 * (new_chi - mu) * (new_chi - mu) / var;
        
        // Compute the Metropolis ratio
        double ratio = logdens_prop - logdens[i];
        ratio = (ratio < 0.0) ? ratio : 0.0;
        ratio = exp(ratio);
        
        // Now randomly decide whether to accept or reject
        double unif_draw = hiprand_uniform_double(&localState);
        
        if (unif_draw < ratio) {
            // Accept this proposal, so save this parameter value and conditional log-posterior
            chi[i] = new_chi;
            logdens[i] = logdens_prop;
        }

        // Copy state back to global memory
        state[i] = localState;

        // Finally, adapt the scale of the proposal distribution
        // TODO: check that the ratio is finite before doing this step
        double decay_sequence = 1.0 / pow(current_iter, c_decay_rate);
        jump_sigma[i] *= exp(decay_sequence / 2.0 * (ratio - c_target_rate));
	}
}

struct zsqr {
    double* theta;
    int p;
    zsqr(double* t, int tdim) : theta(t), p(tdim) {}
    
    __device__ __host__
    double operator()(double chi) {
        double chi_cent = chi - theta[0];
        double chisqr = -0.5 * chi_cent * chi_cent / theta[1];
        return chisqr;
    }
};


int main(void)
{
	int n = 2000; // # of data points
	int m = 1; // # of features per data point (e.g., # of points in the SED for the i^th source)
    int p = 1; // * of characteristics per data point

    // Random number generator and distribution needed to simulate some data
	unsigned int seed = 98724732;
	static thrust::minstd_rand rng(seed);
	thrust::random::experimental::normal_distribution<double> snorm(0., 1.0);
    thrust::random::uniform_real_distribution<double> uniform(0.0, 1.0);
    
    // Population level parameters: theta, where theta parameterizes the distribution of chi
    int dim_theta = 2;
	double mu_popn_true = 20.;  // Average value of the chi values
	double sigma_popn_true = 1.;  // Standard deviation of the chi values
    thrust::host_vector<double> h_theta(dim_theta);  // Allocate memory on host
    h_theta[0] = mu_popn_true;
    h_theta[1] = sigma_popn_true * sigma_popn_true;  // theta = (mu,var)
    
    // Allocate memory for arrays on host
    thrust::host_vector<double> h_meas(n * m);  // The measurements, m values for each of n data points
	thrust::host_vector<double> h_meas_unc(n * m);  // The measurement uncertainties
    thrust::host_vector<double> h_chi(n * p);  // Unknown characteristics, p values for each of n data points
    
    // Scale of proposal distribution for each data point, used by the Metropolis algorithm. When p > 1 this will be
    // the Cholesky factor of the proposal covariance matrix.
    thrust::host_vector<double> h_jump_sigma(n);
    
	// Create simulated characteristics and data
    std::vector<double> true_chi(n * p);
    double sigma_msmt = 3.;  // Standard deviation for the measurement errors
    
	for (int i=0; i<n; i++) {
		for (int j=0; j<p; j++) {
            // First generate true value of the characteristics
			true_chi[i * p + j] = mu_popn_true + sigma_popn_true * snorm(rng);
            // Initialize the scale of the chi proposal distributions to just be the measurement uncertainty
            h_jump_sigma[i * m + j] = sigma_msmt;
        }
        for (int k=0; k<m; k++) {
            // Now generate measurements given the true characteristics
			h_meas_unc[i * m + k] = sigma_msmt;
            // Just assume E(meas|chi) = chi for now
            h_meas[i * m + k] = true_chi[i * p + k] + sigma_msmt * snorm(rng);
		}
	}
    
    // Allocate memory for arrays on device and copy the values from the host
    thrust::device_vector<double> d_meas = h_meas;
    thrust::device_vector<double> d_meas_unc = h_meas_unc;
    thrust::device_vector<double> d_chi = h_chi;
    thrust::device_vector<double> d_jump_sigma = h_jump_sigma;
    thrust::device_vector<double> d_logdens(n);  // log posteriors for an individual data point
    
    thrust::fill(d_logdens.begin(), d_logdens.end(), -1.0e300);
    
	// Load a single set of thetas into constant memory
    double* p_theta = thrust::raw_pointer_cast(&h_theta[0]);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_theta), p_theta, h_theta.size() * sizeof(*p_theta)));
    
    // Cuda grid launch
    dim3 nThreads(256);
    dim3 nBlocks((n + nThreads.x-1) / nThreads.x);
    printf("nBlocks: %d\n", nBlocks.x);  // no more than 64k blocks!
    if (nBlocks.x > 65535)
    {
        std::cerr << "ERROR: Block is too large" << std::endl;
        return 2;
    }

    hiprandState* devStates;  // Create state object for random number generators on the GPU
    // Allocate memory on GPU for RNG states
    CUDA_CALL(hipMalloc((void **)&devStates, nThreads.x * nBlocks.x *
                         sizeof(hiprandState)));
    // Initialize the random number generator states on the GPU
    setup_kernel<<<nBlocks,nThreads>>>(devStates);

    // Wait until everything is done running on the GPU, make sure everything went OK
    CUDA_CALL(hipDeviceSynchronize());

    // Now run the MCMC sampler

    std::ofstream chifile("chis.dat");
    std::ofstream thetafile("thetas.dat");
    
    int mcmc_iter = 1000;
    int naccept_theta = 0;
    for (int i=0; i<mcmc_iter; i++) {
        // Now grab the pointers to the vectors, needed to run the kernel since it doesn't understand Thrust
        // We do this here because the thrust vector are smart, and we want to make sure they don't reassign
        // memory for whatever reason. This is very cheap to do, so better safe than sorry.
        double* p_meas = thrust::raw_pointer_cast(&d_meas[0]);
        double* p_meas_unc = thrust::raw_pointer_cast(&d_meas_unc[0]);
        double* p_chi = thrust::raw_pointer_cast(&d_chi[0]);
        double* p_jump_sigma = thrust::raw_pointer_cast(&d_jump_sigma[0]);
        double* p_logdens = thrust::raw_pointer_cast(&d_logdens[0]);
        int current_iter = i + 1;
        update_chi<<<nBlocks,nThreads>>>(p_chi, p_meas, p_meas_unc, n, p_logdens, p_jump_sigma, devStates, current_iter);
        
        // Generate new theta in parallel with GPU calculation above
        double proposed_theta[2];
        proposed_theta[0] = h_theta[0] + 0.01 * snorm(rng);
        proposed_theta[1] = h_theta[1] + 0.01 * snorm(rng);
        
        CUDA_CALL(hipDeviceSynchronize());
        
        double logdens_pop = thrust::transform_reduce(d_chi.begin(), d_chi.end(), zsqr(proposed_theta,2), 0.0, thrust::plus<double>());
        logdens_pop += -n / 2.0 * proposed_theta[1];
        
        double logdens_old = thrust::reduce(d_logdens.begin(), d_logdens.end());
        
        double lograt = logdens_pop - logdens_old;
        lograt = std::min(lograt, 0.0);
        double ratio = exp(lograt);
        double unif = uniform(rng);
        
        if (unif < ratio) {
            h_theta[0] = proposed_theta[0];
            h_theta[1] = proposed_theta[1];
            CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_theta), p_theta, h_theta.size() * sizeof(*p_theta)));
            naccept_theta++;
        }
        
        thetafile << h_theta[0] << " " << h_theta[1] << std::endl;

        std::cout << current_iter << std::endl;
        thrust::copy(d_chi.begin(), d_chi.end(), h_chi.begin());
        for (int j=0; j<n; j++){
            chifile << " " << h_chi[j];
        }
        chifile << std::endl;
    }
    std::cout << "Number of accepted thetas: " << naccept_theta << std::endl;
    chifile.close();
    thetafile.close();
    hipFree(devStates);  // Free up the memory on the GPU from the RNG states
    
	return 0;
}
