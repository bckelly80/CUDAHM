#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random.h>

#include <stdio.h>

//#include <thrust/sort.h>
//#include <thrust/copy.h>

//#include <thrust/transform_reduce.h>
//#include <thrust/functional.h>

//#include <algorithm>
//#include <cstdlib>

#include <vector>
/*
 typedef thrust::device_vector<double> dvec;
 typedef std::vector<dvec> vdvec;
 
 struct wrapvec
 {
 vdvec v;
 
 wrapvec(int m, int n) : v(m)
 {
 for (int i=0; i<m; i++)	{
 v[i].reserve(n);
 }
 }
 
 double** ptrs()
 {
 thrust::host_vector<double*> h_ptr(v.size());
 for (unsigned int i=0; i<v.size(); i++)
 h_ptr[i] = (double*) thrust::raw_pointer_cast(&(v[i][0]));
 
 thrust::device_vector<double*> d_ptr = h_ptr;
 return (double**) thrust::raw_pointer_cast(&d_ptr[0]);
 }
 };
 */

// Uniform random number transform, used for doing MHA acceptance step on the GPU
static __constant__ thrust::uniform_real_distribution<double> unif(0.0,1.0);

// Norm random number transform, used for generating new parameter proposals on the GPU
static __constant__ thrust::random::experimental::normal_distribution<double> snorm(0.0, 1.0);

// Target acceptance rate for Robust Adaptive Metropolis (RAM).
static __constant__ double target_rate = 0.4;

// Decay rate for proposal scaling factor updates. The proposal scaling factors decay as 1 / niter^decay_rate.
// This is gamma in the notation of Vihola (2012)
static __constant__ double decay_rate = 2.0 / 3.0;

// Current iteration of the MCMC sampler, needed to calculate the decay sequence for the RAM algorithm. Is it
// best to put this in constant memory on the GPU and update from the CPU every iteration?
static __constant__ int current_iter = 0


// Perform the MHA update on the n parameters, done in parallel on the GPU. This is what the kernel does.
__global__
void update_parameters(double* theta, double* parameters, double* data, double* sigmas, int n, double* logdens,
                       double* jump_sigma)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<n)
	{
		double mu = theta[0];  // Get population parameters
		double var = theta[1];
        
        // Propose a new value for the parameters for this data point
        new_parameter = parameters[i] + jump_sigma[i] * snorm(rng);  // Not quite sure how to do this in parallel...
        
        // Compute the conditional log-posterior of the proposed parameter values for this data point
        double logdens_prop = -0.5 * (data[i] - new_parameter) * (data[i] - new_parameter) / (sigmas[i] * sigmas[i]) +
            0.5 * log(var) - 0.5 * (new_parameter - mu) * (new_parameter - mu) / var;
        
        // Compute the Metropolis ratio
        double ratio = logdens_prop - logdens[i];
        ratio = (ratio < 0.0) ? ratio : 0.0;  // Isn't there a min() function I can use to make this more obvious?
        ratio = exp(ratio);
        
        // Now randomly decide whether to accept or reject
        unif_draw = unif(rng);  // Not quite sure if this is the correct way to do this in parallel on the GPU...
        if (unif_draw < ratio) {
            // Accept this proposal, so save this parameter value and conditional log-posterior
            parameters[i] = new_parameter;
            logdens[i] = logdens_prop;
        }
        
        // Finally, adapt the scale of the proposal distribution
        // TODO: check that the ratio is finite before doing this step
        double decay_sequence = 1.0 / pow(niter, decay_rate)
        sigmas[i] *= exp(decay_sequence / 2.0 * (ratio - target_rate))
	}
}


int main(void)
{
	// measurements
	int n = 2000000; // # of items
	int m = 1; // # of features
	/*
     wrapvec d_features(m,n);
     wrapvec d_sigmas(m,n);
     */
	thrust::host_vector<double> h_features(n*m);
	thrust::host_vector<double> h_sigmas(n*m);
    
	unsigned int seed = 98724732;
	double sigma_msmt = 3.;
	static thrust::minstd_rand rng(seed);
	thrust::random::experimental::normal_distribution<double> dist(0., sigma_msmt);
    
	double mu_popn_true = 20.;
	double sigma_popn_true = 1.;
	// Create simulated data; copies to GPU dumbly!
	for (int i=0; i<n; i++) {
		for (int j=0; j<m; j++) {
			h_features[i*m+0] = mu_popn_true + dist(rng);
			h_sigmas[i*m+1] = sigma_popn_true;
		}
	}
	thrust::device_vector<double> d_features = h_features;
	thrust::device_vector<double> d_sigmas = h_sigmas;
    
	// Create array of hyperparameter values.
	// Here we condition on sigma_popn_true (for simple analytical result).
	// Currently ineffecient; should build host vector and copy over.
	int dim_theta = 2;
	int n_theta = 11;
	double mu_lo = mu_popn_true - 2*sigma_msmt/sqrt(n);
	double mu_hi = mu_popn_true + 2*sigma_msmt/sqrt(n);
	double dmu = (mu_hi - mu_lo)/(n_theta-1.);
	double mu;
	thrust::host_vector<double> h_theta(dim_theta*n_theta);
	for (int i=0; i<n_theta; i++) {
		mu = mu_lo + i*dmu;
		h_theta[i*dim_theta] = mu;
		h_theta[i*dim_theta+1] = sigma_popn_true;
	}
	thrust::device_vector<double> d_theta = h_theta;
    
	// To load a single set of thetas into constant memory, copy to a global:
	// hipMemcpyToSymbol(HIP_SYMBOL(c_theta), p_theta, d_theta.size() * sizeof(*p_theta));
    
	// Alloc mem for marginals for individuals, for each set of hyperparams.
	thrust::device_vector<double> d_marg(n*n_theta);
    
	{
		// log marginal likelhoods in parallel on all threads independently
		double* p_marg = thrust::raw_pointer_cast(&d_marg[0]);
		double* p_theta = thrust::raw_pointer_cast(&d_theta[0]);
		double* p_features = thrust::raw_pointer_cast(&d_features[0]);
		double* p_sigmas = thrust::raw_pointer_cast(&d_sigmas[0]);
        
		// cuda grid launch
		dim3 nThreads(32,8);
		dim3 nBlocks((n + nThreads.x-1) / nThreads.x, (n_theta + nThreads.y-1) / nThreads.y);
		printf("nBlocks: %d  %d\n", nBlocks.x, nBlocks.y);  // no more than 64k blocks!
		marginals<<<nBlocks,nThreads>>>(p_theta, dim_theta, n_theta,
                                        p_features, p_sigmas, m, n, p_marg);
		// wait for it to finish
		hipError_t err = hipDeviceSynchronize();
        
		thrust::host_vector<double> h_marg = d_marg;
		for (int i=0; i<20; i++) {
            printf("%d %20.10f \n", i, h_marg[i]);
		}
        
		// Loop over hyperparams; reduce over individuals for each case.
		for (int i=0; i<n_theta; i++) {
			int start = i*n;
			int end = start + n;
			double log_marg = 0;
			log_marg = thrust::reduce(d_marg.begin()+start, d_marg.begin()+end);
			//std::cout << i << " " << log_marg << std::endl;
            printf("%d %20.10f %20.10f \n", i, log_marg, h_theta[i*dim_theta]);
		}
        
	}
    
	return 0;
    
}
